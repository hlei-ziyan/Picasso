
#include <hip/hip_runtime.h>
#ifndef M_PI
#define M_PI           3.14159265358979323846F  /* pi */
#endif

struct point3d
{
    float x=0, y=0, z=0;
};

// database:  B*N*3
// query:     B*M*3
// nnIndex:   B*M*nnSample
// nnCount:   B*M
// nnDist:    B*M*nnSample
__global__ void cal_nnidx_sphere(int B, int N, int M, int nnSample, float radius,
                              const float* database, const float* query,
                              int* nnIndex, int* nnCount, float* nnDist)
{
    // get the neighbor indices
    point3d ptQuery, pt, delta;
    for(int i=blockIdx.x;i<B;i+=gridDim.x)
    {
        for(int j=threadIdx.x;j<M;j+=blockDim.x)
        {
            ptQuery.x = query[i*M*3+j*3];
            ptQuery.y = query[i*M*3+j*3+1];
            ptQuery.z = query[i*M*3+j*3+2];

            int s=0; // to count the number of neighbors
            while(s==0) //require a minimum of 1 neighbor point
            {
                //re-initialziation
                s = 0;

                for(int k=0;k<N;k++)
                {
                    pt.x = database[i*N*3+k*3];
                    pt.y = database[i*N*3+k*3+1];
                    pt.z = database[i*N*3+k*3+2];

                    delta.x = pt.x - ptQuery.x;
                    delta.y = pt.y - ptQuery.y;
                    delta.z = pt.z - ptQuery.z;

                    float dist3D = delta.x*delta.x + delta.y*delta.y + delta.z*delta.z; // squared 3D
                    dist3D = sqrtf(dist3D); //sqrt

                    if (dist3D<radius && fabs(dist3D-radius)>1e-6) // find a neighbor in range
                    {
                        if (s<nnSample) // sample NO=nnSample neighbor points, requires shuffling of points order in every epoch
                        {
                            nnIndex[i*M*nnSample+j*nnSample+s] = k;
                            nnDist[i*M*nnSample+j*nnSample+s] = dist3D; // sqrt, not the squared one
                        }
                        s++;
                    }
                }
                radius += 0.05;
            }

            nnCount[i*M+j] = s<nnSample?s:nnSample;
        }
    }
}


// database:  B*N*3
// query:    B*M*3
// nnIndex:  B*M*nnSample*2
// nnCount:  B*M
__global__ void cal_nnidx_cube(int B, int N, int M, int gridSize, int nnSample, float length,
                              const float* database, const float* query, int* nnIndex, int* nnCount)
{
    // get the neighbor indices, and compute their indices in the filter/kernel bins
    point3d ptQuery, pt, delta;
    for(int i=blockIdx.x;i<B;i+=gridDim.x)
    {
        for(int j=threadIdx.x;j<M;j+=blockDim.x)
        {
            ptQuery.x = query[i*M*3+j*3];
            ptQuery.y = query[i*M*3+j*3+1];
            ptQuery.z = query[i*M*3+j*3+2];

            int s=0; // to count the number of neighbors
            while(s==0) //require a minimum of 1 neighbor point
            {
                //re-initialziation
                s = 0; // to count the number of neighbors

                for(int k=0;k<N;k++)
                {
                    pt.x = database[i*N*3+k*3];
                    pt.y = database[i*N*3+k*3+1];
                    pt.z = database[i*N*3+k*3+2];

                    delta.x = pt.x - ptQuery.x;
                    delta.y = pt.y - ptQuery.y;
                    delta.z = pt.z - ptQuery.z;

                    if (abs(delta.x)<length/2 && abs(delta.y)<length/2 && abs(delta.z)<length/2 && s<nnSample)
                    {
                        // calculate bin index in the cubic filter/kernel
                        int xId = (delta.x + length/2)/(length/gridSize); //[0, gridSize)
                        int yId = (delta.y + length/2)/(length/gridSize); //[0, gridSize)
                        int zId = (delta.z + length/2)/(length/gridSize); //[0, gridSize)

//                        float dist3D = delta.x*delta.x + delta.y*delta.y + delta.z*delta.z; // squared 3D
//                        int binID;
//                        if (dist3D<1e-6)
//                            binID = 0;
//                        else
//                            binID = xId*gridSize*gridSize + yId*gridSize + zId + 1;

                        int binID = xId*gridSize*gridSize + yId*gridSize + zId;

                        nnIndex[i*M*nnSample*2+j*nnSample*2+s*2] = k;
                        nnIndex[i*M*nnSample*2+j*nnSample*2+s*2+1] = binID;
                        s++;
                    }
                }
                length += 0.1;
            }
            nnCount[i*M+j] = s<nnSample?s:nnSample;
        }
    }
}

/* This part's code is based on the 3nn weighted interpolation of PointNet++
   of Charles R. Qi.
*/
// database:  B*N*3
// query:     B*M*3
// nnIndex:   B*M*nnOut
// nnDist:    B*M*nnOut
__global__ void cal_nnidx(int B, int N, int M, const float* database,
                          const float* query, int* nnIndex, float* nnDist)
{
    // get the neighbor indices
    point3d ptQuery, pt, delta;
    for(int i=blockIdx.x;i<B;i+=gridDim.x)
    {
        for(int j=threadIdx.x;j<M;j+=blockDim.x)
        {
            ptQuery.x = query[i*M*3+j*3];
            ptQuery.y = query[i*M*3+j*3+1];
            ptQuery.z = query[i*M*3+j*3+2];

            float best1=1e40, best2=1e40, best3=1e40, best4=1e40;
            int   besti1=0, besti2=0, besti3=0, besti4=0;

            for(int k=0;k<N;k++)
            {
                pt.x = database[i*N*3+k*3];
                pt.y = database[i*N*3+k*3+1];
                pt.z = database[i*N*3+k*3+2];

                delta.x = pt.x - ptQuery.x;
                delta.y = pt.y - ptQuery.y;
                delta.z = pt.z - ptQuery.z;

                float dist3D = delta.x*delta.x + delta.y*delta.y + delta.z*delta.z; // squared 3D

                if (dist3D<best1) { // This is from PointNet++
                    best4=best3;
                    besti4=besti3;
                    best3=best2;
                    besti3=besti2;
                    best2=best1;
                    besti2=besti1;
                    best1=dist3D;
                    besti1=k;
                } else if (dist3D<best2) {
                    best4=best3;
                    besti4=besti3;
                    best3=best2;
                    besti3=besti2;
                    best2=dist3D;
                    besti2=k;
                } else if (dist3D<best3) {
                    best4=best3;
                    besti4=besti3;
                    best3=dist3D;
                    besti3=k;
                } else if (dist3D<best4) {
                    best4=dist3D;
                    besti4=k;
                }

                nnIndex[i*M*3+j*3]   = besti1;
                nnIndex[i*M*3+j*3+1] = besti2;
                nnIndex[i*M*3+j*3+2] = besti3;
                nnIndex[i*M*3+j*3+3] = besti4;
                nnDist[i*M*3+j*3]    = sqrtf(best1); // sqrt, not the squared one
                nnDist[i*M*3+j*3+1]  = sqrtf(best2); // sqrt, not the squared one
                nnDist[i*M*3+j*3+2]  = sqrtf(best3); // sqrt, not the squared one
                nnDist[i*M*3+j*3+3]  = sqrtf(best4); // sqrt, not the squared one
            }
        }
    }
}

void buildSphereNeighborLauncher(int B, int N, int M, int nnSample, float radius,
                                 const float* database, const float* query, int* nnIndex,
                                 int* nnCount, float* nnDist)
{
    cal_nnidx_sphere<<<B,1024>>>(B, N, M, nnSample, radius,
                               database, query, nnIndex, nnCount, nnDist);
}

void buildCubeNeighborLauncher(int B, int N, int M, int gridSize, int nnSample, float length,
                                 const float* database, const float* query, int* nnIndex, int* nnCount)
{
    cal_nnidx_cube<<<B,1024>>>(B, N, M, gridSize, nnSample, length, database, query, nnIndex, nnCount);
}

void buildNearestNeighborLauncher(int B, int N, int M, const float* database,
                                  const float* query, int* nnIndex, float* nnDist)
{
    cal_nnidx<<<B,1024>>>(B, N, M, database, query, nnIndex, nnDist);
}